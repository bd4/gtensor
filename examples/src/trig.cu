#include <math.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

__global__ void kernel_add_sq(float* c, const float* a, const float* b, int N)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < N) {
    c[i] = a[i] * a[i] + b[i] * b[i];
  }
}

inline hipError_t CHECK(hipError_t err)
{
  if (err != hipSuccess) {
    printf("Error: %d %s\n", err, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  return err;
}

int main(int argc, char** argv)
{
  const int N = 1024 * 1024;
  const int block_size = 256;
  int i;
  int size = N * sizeof(float);

  float* h_a = (float*)malloc(size);
  float* h_b = (float*)malloc(size);
  float* h_c = (float*)malloc(size);

  for (i = 0; i < N; i++) {
    h_a[i] = sin(i);
    h_b[i] = cos(i);
  }

  float *d_a, *d_b, *d_c;
  CHECK(hipMalloc((void**)&d_a, size));
  CHECK(hipMalloc((void**)&d_b, size));
  CHECK(hipMalloc((void**)&d_c, size));

  CHECK(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice));

  // assumes block_size devices N
  kernel_add_sq<<<N / block_size, block_size>>>(d_c, d_a, d_b, N);

  CHECK(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost));

  printf("=== sin(i) + cos(i)\n");
  for (i = 0; i < N; i += N / 32) {
    printf("%0.2f = %0.2f + %0.2f\n", h_c[i], h_a[i], h_b[i]);
  }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(h_a);
  free(h_b);
  free(h_c);
}
